﻿#ifndef _INC_ARRAY_H_
#define _INC_ARRAY_H_
//"C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v4.2\bin\nvcc.exe"  --cl-version 2010 --cubin %1CUDAFunctions.cu -ccbin "C:\Program Files (x86)\Microsoft Visual Studio 10.0\VC\bin" -I "C:\Program Files (x86)\Microsoft Visual Studio 10.0\VC\include" -m64 -arch=sm_21 -o "%1Properties\Resources\CUDAFunctions.21.x64.cubin"

#define byte unsigned char
#define ushort unsigned short

struct LayerInfo
{
	int nWidthDiff;
	int nForegroundStart;
	int nBackgroundStart;
	int nBackgroundStop;
	int nCropLeft;
	int nCropRight;
	int nWidth;
	int nAlphaConstant;
	int nCropWidth;
	int nCropHeight;
	float nShiftPosition;
	int bShiftVertical;
    int nAlphaType;
	int nOffsetLeft; 
	int nOffsetTop;
	float nShiftTotal;
};
struct MergeInfo
{
	int nLayersQty;
	int nBackgroundSize;
	int nBackgroundWidth;
	int nBackgroundAlphaType;
	LayerInfo aLayerInfos[];
};

extern "C" __global__ void CUDAFrameMerge(byte **pLayers, MergeInfo *pMergeInfo, byte *pAlphaMap)
{
	int nBGIndxPixel = blockIdx.x * blockDim.x + threadIdx.x; //это п. номер только по одному цвету и по BG !!

	if (nBGIndxPixel < pMergeInfo->nBackgroundSize) //2-й - это размер BG, 3-й - ширина BG, 4-й - делать ли задник? 5-й - инфа про FG1; 
	{									//Периодичность PRECOMPUTED_INFO_PERIOD - 1-й. 0-й - это количество слоёв
		int M, nIndxIndent, nRow;
		int nBGIndxRed, nBGIndxGreen, nBGIndxBlue, nBGIndxAlpha, nFGIndx;
		byte nFGColorRed, nFGColorGreen, nFGColorBlue, nFGColorAlpha;
		int nNextIndxRed, nNextIndxGreen, nNextIndxBlue, nNextIndxAlpha, nPixelAlphaIndx;	
		byte nPixelAlpha;
        int nMaskIndx = -1;
		LayerInfo* pLayerInfo;

		M = nBGIndxPixel / pMergeInfo->nBackgroundWidth; //M=(int)(BI/BW) т.е. с отбрасыванием дробной части.
		nIndxIndent = nBGIndxPixel - M * pMergeInfo->nBackgroundWidth;

		nBGIndxRed = nBGIndxPixel * 4;
		nBGIndxGreen = nBGIndxRed + 1;
		nBGIndxBlue = nBGIndxRed + 2;
		nBGIndxAlpha = nBGIndxRed + 3;
		pLayers[0][nBGIndxRed] = 0;
		pLayers[0][nBGIndxGreen] = 0;
		pLayers[0][nBGIndxBlue] = 0;
        if(1 == pMergeInfo->nBackgroundAlphaType)
            nMaskIndx = nBGIndxAlpha;
        else
            pLayers[0][nBGIndxAlpha] = pMergeInfo->nBackgroundAlphaType;

		for (ushort nLayerIndx = 1; pMergeInfo->nLayersQty > nLayerIndx; nLayerIndx++)
		{ 
			pLayerInfo = &pMergeInfo->aLayerInfos[(int)(nLayerIndx - 1)];
			if ((nBGIndxPixel >= pLayerInfo->nBackgroundStart) && (nBGIndxPixel <= pLayerInfo->nBackgroundStop) && (nIndxIndent >= pLayerInfo->nCropLeft) && (nIndxIndent <= pLayerInfo->nCropRight))
			{
				nFGIndx = (nBGIndxPixel + M * pLayerInfo->nWidthDiff - pLayerInfo->nForegroundStart) * 4;  
																								//формулу см. в методе Intersect.
                if (1 == pLayerInfo->nAlphaType) //леер является маской
                {
                    nMaskIndx = nFGIndx + 3;
                    continue;
                }
				nFGColorAlpha = pLayers[nLayerIndx][nFGIndx + 3];
                if (-1 < nMaskIndx) //применяем маску
                {
                    if (255 == pLayers[nLayerIndx - 1][nMaskIndx]) //отрезали пиксел по маске
                    {
                        nMaskIndx = -1;
                        continue;
                    }
                    nFGColorAlpha = (byte)(255.5 - (float)nFGColorAlpha * pLayers[nLayerIndx - 1][nMaskIndx] / 255);
                    nMaskIndx = -1;
                }
				nFGColorRed = pLayers[nLayerIndx][nFGIndx];
				nFGColorGreen = pLayers[nLayerIndx][nFGIndx + 1];
				nFGColorBlue = pLayers[nLayerIndx][nFGIndx + 2];
				
                if (0 == pLayerInfo->nAlphaType)
                {
					if (0 != pLayerInfo->nShiftPosition || 0 != pLayerInfo->nShiftTotal)  // && 1 > pLayerInfo->nShiftPosition && -1 < pLayerInfo->nShiftPosition
					{
						if (pLayerInfo->bShiftVertical)
						{
							if (0 < pLayerInfo->nShiftPosition)
							{
								nPixelAlpha = nFGColorAlpha;
								nFGColorAlpha = (byte)((nFGColorAlpha + 1) * (1 - pLayerInfo->nShiftPosition));
								nRow = M - (pLayerInfo->nBackgroundStart / pMergeInfo->nBackgroundWidth);
								if (nRow < (pLayerInfo->nCropHeight - 1))
								{
									nNextIndxRed = nFGIndx + (pLayerInfo->nWidth * 4);
									nNextIndxGreen = nNextIndxRed + 1;
									nNextIndxBlue = nNextIndxRed + 2;
									nNextIndxAlpha = nNextIndxRed + 3;
									if (0 < pLayers[nLayerIndx][nNextIndxAlpha])
									{
										if (0 < (nPixelAlpha = (byte)((pLayers[nLayerIndx][nNextIndxAlpha] + 1) * pLayerInfo->nShiftPosition)))
										{
											if (0 == nFGColorAlpha || 254 < nPixelAlpha)
											{
												nFGColorRed = pLayers[nLayerIndx][nNextIndxRed];
												nFGColorGreen = pLayers[nLayerIndx][nNextIndxGreen];
												nFGColorBlue = pLayers[nLayerIndx][nNextIndxBlue];
											}
											else
											{
												nPixelAlphaIndx = (nPixelAlpha - 1) * 65536;
												nFGColorRed = pAlphaMap[nPixelAlphaIndx + (256 * nFGColorRed) + pLayers[nLayerIndx][nNextIndxRed]];
												nFGColorGreen = pAlphaMap[nPixelAlphaIndx + (256 * nFGColorGreen) + pLayers[nLayerIndx][nNextIndxGreen]];
												nFGColorBlue = pAlphaMap[nPixelAlphaIndx + (256 * nFGColorBlue) + pLayers[nLayerIndx][nNextIndxBlue]];
											}
										}
										if (255 < nFGColorAlpha + nPixelAlpha)
											nFGColorAlpha = 255;
										else
											nFGColorAlpha += nPixelAlpha;
									}
								}
							}
							else
							{
							}
						}
						else
						{
							if (0 < pLayerInfo->nShiftPosition)
							{
							
							}
							else
							{
							}
						}
					}
				
					nPixelAlpha = pLayerInfo->nAlphaConstant;

					if (255 == nPixelAlpha)
						nPixelAlpha = nFGColorAlpha;
					else if (0 == nFGColorAlpha)
						nPixelAlpha=0;
					else if (0 < nPixelAlpha && 255 > nFGColorAlpha)                        // объединение альфы слоя с константной альфой !!!!
						nPixelAlpha = (byte)((float)nFGColorAlpha * nPixelAlpha / 255 + 0.5);
				}
                else
                    nPixelAlpha = 255;
					
				if (0 < nPixelAlpha)
				{
					if (255 == nPixelAlpha || 0 == pLayers[0][nBGIndxAlpha])
					{
						pLayers[0][nBGIndxRed] = nFGColorRed;
						pLayers[0][nBGIndxGreen] = nFGColorGreen;
						pLayers[0][nBGIndxBlue] = nFGColorBlue;
					}
					else
					{							//индекс меньше, т.к. 0-е значение альфы мы не считаем и все индексы сдвинулись...
						nPixelAlphaIndx = (nPixelAlpha - 1) * 65536;
						pLayers[0][nBGIndxRed] = pAlphaMap[nPixelAlphaIndx + (256 * pLayers[0][nBGIndxRed]) + nFGColorRed];
						pLayers[0][nBGIndxGreen] = pAlphaMap[nPixelAlphaIndx + (256 * pLayers[0][nBGIndxGreen]) + nFGColorGreen];
						pLayers[0][nBGIndxBlue] = pAlphaMap[nPixelAlphaIndx + (256 * pLayers[0][nBGIndxBlue]) + nFGColorBlue];
					}
					if (pLayers[0][nBGIndxAlpha] < nPixelAlpha)   // очередная попытка примирить альфу с действительностью ))
						pLayers[0][nBGIndxAlpha] = nPixelAlpha;
				}
			}
		}
	}
}
#endif // _INC_ARRAY_H_
