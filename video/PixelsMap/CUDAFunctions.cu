﻿#ifndef _INC_ARRAY_H_
#define _INC_ARRAY_H_
//"C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v4.2\bin\nvcc.exe"  --cl-version 2010 --cubin %1CUDAFunctions.cu -ccbin "C:\Program Files (x86)\Microsoft Visual Studio 10.0\VC\bin" -I "C:\Program Files (x86)\Microsoft Visual Studio 10.0\VC\include" -m64 -arch=sm_21 -o "%1Properties\Resources\CUDAFunctions.21.x64.cubin"
#define INT_MAX       2147483647    /* maximum (signed) int value */
#define byte unsigned char
#define ushort unsigned short
//#include <stdlib.h>

struct LayerInfo
{
	int nCropTopLineInBG;
	int nCropBottomLineInBG;
	int nCropLeft_4;
	int nWidth_4;
	int nCropWidth_4;
	int nLeft_4;
	int nTop;
	int nAlphaConstant;
	int nHalfDeltaPxX_4;
	int nHalfDeltaPxY_4;
	int nHalfPathShiftPositionByteX;
	int nHalfPathShiftPositionByteY;
	int nShiftPositionByteX;
	int nShiftPositionByteY;
	int nAlphaType;
	int nOffsetTop;
	float nShiftTotalX;
	int nBytesQty;
};
struct MergeInfo
{
	int nLayersQty;
	int nBackgroundSize;
	int nBackgroundWidth_4;
	int nBackgroundAlphaType;
	int nBackgroundHight;
	int nBackgroundWidth;
	LayerInfo aLayerInfos[];
};

extern "C" __global__ void CUDAFrameMerge(byte **pLayers, MergeInfo *pMergeInfo, byte *pAlphaMap, int *pAlphaMap_info3d, ushort *pAlphaMap_info2d, byte *pAlphaMap2, ushort *pAlphaMap2_info2d, byte *pAlphaMap3, ushort *pAlphaMap3_info2d)
{
	int nBGIndxLine = blockIdx.y * blockDim.y + threadIdx.y;  // the first block is 0
	int nBGIndxRed = blockIdx.x * blockDim.x + threadIdx.x;  // the first block is 0
	if (nBGIndxLine >= pMergeInfo->nBackgroundHight || nBGIndxRed >= pMergeInfo->nBackgroundWidth)
		return;

	int nBGIndxPixelStartRed = nBGIndxLine * pMergeInfo->nBackgroundWidth_4;
	nBGIndxRed = nBGIndxPixelStartRed + 4 * nBGIndxRed;

	struct LayerInfo* cLayerInfo;
	int nBGIndxGreen, nBGIndxBlue, nBGIndxAlpha, nFGIndxRed;
	byte nFGColorRed, nFGColorGreen, nFGColorBlue, nFGColorAlpha, nExtMaskAlpha;
	int nPixelAlphaIndx;
	byte nPixelAlpha, nMaskAllUpper, nMaskCurrent;
	ushort nLayerIndx;
	bool bMainCondition, bFieldsCondition = false;
	bool bElse;     // нужна для экономии, чтоб не каждый раз присваивать, а только когда надо
	bool bBGChanged;        // тоже чтобы менять, только если не менялся фон
	bool bBGAChanged;        // тоже чтобы менять, только если не менялся фон
	int nRedPlusDelta;
	int nUpPxIndx;
	int nRowBeginingIndx;
	int nLeftPxIndx;
	int nRowEndingIndx;

	// ex cLLI  (struct LineLayerInfo)
	int nBGCropStartRed; // before main if     (+)
	int nFGCropStartRed; // in main if (1 time)   (*)
	int nBGCropEndRed;  // before main if    (+ -)
	bool bRowUpper = false;  // for shift      (- -)
	int nBgFgLinesDelta;   // for fields (1 time)   (-)
	int nFGLineBeginningRed; // for fields and shift  (* /)
							 // ---------

	if (255 == pMergeInfo->nBackgroundAlphaType || 0 == pMergeInfo->nBackgroundAlphaType)
	{
		pLayers[0][nBGIndxRed + 3] = pMergeInfo->nBackgroundAlphaType;
		bBGAChanged = true;
	}
	else
		bBGAChanged = false;

	nMaskAllUpper = 0;
	nExtMaskAlpha = 0;
	bBGChanged = false;

	for (nLayerIndx = 1; pMergeInfo->nLayersQty > nLayerIndx; nLayerIndx++)
	{
		cLayerInfo = &pMergeInfo->aLayerInfos[(int)(nLayerIndx - 1)];

		// ex cLLI init
		if (nBGIndxLine < cLayerInfo->nCropTopLineInBG || nBGIndxLine > cLayerInfo->nCropBottomLineInBG)
		{
			nBGCropStartRed = INT_MAX;   // not main_if
			nFGCropStartRed = INT_MIN;
			nBGCropEndRed = INT_MIN;
		}
		else
		{
			nBGCropStartRed = nBGIndxPixelStartRed + cLayerInfo->nCropLeft_4;
			nBGCropEndRed = nBGCropStartRed + cLayerInfo->nCropWidth_4 - 4;
		}
		// -----------

		bMainCondition = nBGIndxRed >= nBGCropStartRed && nBGIndxRed <= nBGCropEndRed;

		if (cLayerInfo->nAlphaType > 0 && cLayerInfo->nAlphaType < 5)  // т.е. маска типа 1,2,3,4
		{
			if (nExtMaskAlpha < 255 || (cLayerInfo->nAlphaType == 3 || cLayerInfo->nAlphaType == 4) && nMaskAllUpper < 255)
			{
				if (bMainCondition)
				{
					// ex cLLI init
					if (cLayerInfo->nCropTopLineInBG == 0)
						nFGCropStartRed = (nBGIndxLine - cLayerInfo->nTop) * cLayerInfo->nWidth_4;
					else
						nFGCropStartRed = (nBGIndxLine - cLayerInfo->nCropTopLineInBG) * cLayerInfo->nWidth_4;
					if (nBGIndxPixelStartRed >= nBGCropStartRed)
						nFGCropStartRed -= cLayerInfo->nLeft_4;
					// -----------

					nMaskCurrent = pLayers[nLayerIndx][(nBGIndxRed - nBGCropStartRed + nFGCropStartRed) + 3];
				}
				else
					nMaskCurrent = 0;    // т.е. маски нет тут.        // лаер является маской   и там, где слоя вообще нет, то всё-равно отрезаем, т.к. на то она и маска, что где картинки нет, то считаем это всё альфой

				if (cLayerInfo->nAlphaType == 2 || cLayerInfo->nAlphaType == 4) // т.е. инвертированная маска
					nMaskCurrent = (byte)(255 - nMaskCurrent);

				if (cLayerInfo->nAlphaType < 3)
					nExtMaskAlpha = (byte)(nExtMaskAlpha + nMaskCurrent > 255 ? 255 : nExtMaskAlpha + nMaskCurrent); // gcc понимает это так же, т.е. (nExtMaskAlpha + nMaskCurrent)

				if (cLayerInfo->nAlphaType == 3 || cLayerInfo->nAlphaType == 4)  // т.е. all upper mask
					nMaskAllUpper = (byte)(nMaskAllUpper + nMaskCurrent > 255 ? 255 : nMaskAllUpper + nMaskCurrent);
			}
			continue;
		}
		nExtMaskAlpha = (byte)(nExtMaskAlpha + nMaskAllUpper > 255 ? 255 : nExtMaskAlpha + nMaskAllUpper);

		if (255 == nExtMaskAlpha) //отрезали пиксел по маске, т.е. наш слой стал A = 0
		{
			nExtMaskAlpha = 0;
			continue;
		}

		bFieldsCondition = false;
		if (!bMainCondition)
		{
			nRedPlusDelta = nBGIndxRed + cLayerInfo->nHalfDeltaPxX_4;
			bFieldsCondition = cLayerInfo->nShiftTotalX != 0 &&    // есть поля в слое
				(cLayerInfo->nHalfDeltaPxX_4 > 0 && nRedPlusDelta >= nBGCropStartRed && nRedPlusDelta <= nBGCropEndRed ||    // движение вправо и можно показать поля за рамками кропа
					cLayerInfo->nHalfDeltaPxX_4 < 0 && nRedPlusDelta <= nBGCropEndRed && nRedPlusDelta >= nBGCropStartRed);    // движение влево и можно показать поля за рамками кропа
		}
		if (bMainCondition || bFieldsCondition)   //main_if
		{
			// ex cLLI init
			if (nBGCropStartRed != INT_MAX)
			{
				if (cLayerInfo->nCropTopLineInBG == 0)
					nFGCropStartRed = (nBGIndxLine - cLayerInfo->nTop) * cLayerInfo->nWidth_4;
				else
					nFGCropStartRed = (nBGIndxLine - cLayerInfo->nCropTopLineInBG) * cLayerInfo->nWidth_4;
				if (nBGIndxPixelStartRed >= nBGCropStartRed)
					nFGCropStartRed -= cLayerInfo->nLeft_4;
			}
			// -----------

			nFGIndxRed = (nBGIndxRed - nBGCropStartRed + nFGCropStartRed);
			bElse = false;

			//if (0 == cLayerInfo.nAlphaType) // т.е. наш слой не альфирующий, а обычный слой с альфой RGBA     // вроде не доёдем же сюда с > 0 ????  надо попробовать убрать // попробовал ))
			//{
			//#region обработка шифтов и движений
			if (0 == cLayerInfo->nShiftTotalX) // поля не нужны и обрабатываем только неточное положение пикселя
			{
				if (0 != cLayerInfo->nShiftPositionByteX || 0 != cLayerInfo->nShiftPositionByteY)
				{
					if (0 != cLayerInfo->nShiftPositionByteX && 0 == cLayerInfo->nShiftPositionByteY)  // попали не точно в пиксель по Х   (берём с левого пикселя)
					{
						//#region не точно по X
						nLeftPxIndx = nFGIndxRed - 4;
						// ex cLLI init
						if (nBGIndxPixelStartRed < nBGCropStartRed)
							nFGLineBeginningRed = nFGCropStartRed;
						else
							nFGLineBeginningRed = ((int)((float)nFGCropStartRed / cLayerInfo->nWidth_4)) * cLayerInfo->nWidth_4;
						// -----------			

						if (nLeftPxIndx >= nFGLineBeginningRed) // левый пиксель ещё в нашей строке
						{
							if (pLayers[nLayerIndx][nFGIndxRed + 3] == 0) // сложение с пустым пикселем несет опасность появления (проверено) чёрного цвета из него.
							{
								nFGColorRed = pLayers[nLayerIndx][nLeftPxIndx];
								nFGColorGreen = pLayers[nLayerIndx][nLeftPxIndx + 1];
								nFGColorBlue = pLayers[nLayerIndx][nLeftPxIndx + 2];
							}
							else if (pLayers[nLayerIndx][nLeftPxIndx + 3] == 0)
							{
								nFGColorRed = pLayers[nLayerIndx][nFGIndxRed];
								nFGColorGreen = pLayers[nLayerIndx][nFGIndxRed + 1];
								nFGColorBlue = pLayers[nLayerIndx][nFGIndxRed + 2];
							}
							else
							{
								nFGColorRed = pAlphaMap[pAlphaMap_info3d[cLayerInfo->nShiftPositionByteX - 1] + pAlphaMap_info2d[pLayers[nLayerIndx][nFGIndxRed]] + pLayers[nLayerIndx][nLeftPxIndx]];
								nFGColorGreen = pAlphaMap[pAlphaMap_info3d[cLayerInfo->nShiftPositionByteX - 1] + pAlphaMap_info2d[pLayers[nLayerIndx][nFGIndxRed + 1]] + pLayers[nLayerIndx][nLeftPxIndx + 1]];
								nFGColorBlue = pAlphaMap[pAlphaMap_info3d[cLayerInfo->nShiftPositionByteX - 1] + pAlphaMap_info2d[pLayers[nLayerIndx][nFGIndxRed + 2]] + pLayers[nLayerIndx][nLeftPxIndx + 2]];
							}
							nFGColorAlpha = pAlphaMap[pAlphaMap_info3d[cLayerInfo->nShiftPositionByteX - 1] + pAlphaMap_info2d[pLayers[nLayerIndx][nFGIndxRed + 3]] + pLayers[nLayerIndx][nLeftPxIndx + 3]];
						}
						else // если наш пиксель первый в строке - он просто "ослабнет"
						{
							nFGColorRed = pLayers[nLayerIndx][nFGIndxRed];
							nFGColorGreen = pLayers[nLayerIndx][nFGIndxRed + 1];
							nFGColorBlue = pLayers[nLayerIndx][nFGIndxRed + 2];
							nFGColorAlpha = pAlphaMap[pAlphaMap_info3d[cLayerInfo->nShiftPositionByteX - 1] + pAlphaMap_info2d[pLayers[nLayerIndx][nFGIndxRed + 3]]];
						}
						//#endregion
					}
					else if (0 == cLayerInfo->nShiftPositionByteX && 0 != cLayerInfo->nShiftPositionByteY)  // попали не точно в пиксель по Y   (берём с верхнего пикселя)
					{
						//#region не точно по Y
						// ex cLLI init
						bRowUpper = false;
						if (nBGIndxLine - 1 >= cLayerInfo->nCropTopLineInBG && nBGIndxLine - 1 <= cLayerInfo->nCropBottomLineInBG)
							bRowUpper = true;
						// -----------

						if (bRowUpper)  // в нашем FG есть строка выше текущей, которая входит в кроп по BG 
						{
							nUpPxIndx = nFGIndxRed - cLayerInfo->nWidth_4;
							if (pLayers[nLayerIndx][nFGIndxRed + 3] == 0) // сложение с пустым пикселем несет опасность появления (проверено) чёрного цвета из него.
							{
								nFGColorRed = pLayers[nLayerIndx][nUpPxIndx];
								nFGColorGreen = pLayers[nLayerIndx][nUpPxIndx + 1];
								nFGColorBlue = pLayers[nLayerIndx][nUpPxIndx + 2];
							}
							else if (pLayers[nLayerIndx][nUpPxIndx + 3] == 0)
							{
								nFGColorRed = pLayers[nLayerIndx][nFGIndxRed];
								nFGColorGreen = pLayers[nLayerIndx][nFGIndxRed + 1];
								nFGColorBlue = pLayers[nLayerIndx][nFGIndxRed + 2];
							}
							else
							{
								nFGColorRed = pAlphaMap[pAlphaMap_info3d[cLayerInfo->nShiftPositionByteY - 1] + pAlphaMap_info2d[pLayers[nLayerIndx][nFGIndxRed]] + pLayers[nLayerIndx][nUpPxIndx]];
								nFGColorGreen = pAlphaMap[pAlphaMap_info3d[cLayerInfo->nShiftPositionByteY - 1] + pAlphaMap_info2d[pLayers[nLayerIndx][nFGIndxRed + 1]] + pLayers[nLayerIndx][nUpPxIndx + 1]];
								nFGColorBlue = pAlphaMap[pAlphaMap_info3d[cLayerInfo->nShiftPositionByteY - 1] + pAlphaMap_info2d[pLayers[nLayerIndx][nFGIndxRed + 2]] + pLayers[nLayerIndx][nUpPxIndx + 2]];
							}
							nFGColorAlpha = pAlphaMap[pAlphaMap_info3d[cLayerInfo->nShiftPositionByteY - 1] + pAlphaMap_info2d[pLayers[nLayerIndx][nFGIndxRed + 3]] + pLayers[nLayerIndx][nUpPxIndx + 3]];
						}
						else
						{
							nFGColorRed = pLayers[nLayerIndx][nFGIndxRed];
							nFGColorGreen = pLayers[nLayerIndx][nFGIndxRed + 1];
							nFGColorBlue = pLayers[nLayerIndx][nFGIndxRed + 2];
							nFGColorAlpha = pAlphaMap[pAlphaMap_info3d[cLayerInfo->nShiftPositionByteY - 1] + pAlphaMap_info2d[pLayers[nLayerIndx][nFGIndxRed + 3]]];
						}
						//#endregion
					}
					else //  попали не точно в пиксель и по Х и по Y
					{
						bElse = true;
						// не реализовано! не нужно пока-> Но описание вот:
						// обычно берем инфу с двух соседних пикселей в пропорции шифта - с этого и с пикселя слева от этого. Т.к. был перелет из-за отбрасывания дробной части от X (= x-floor(x))  (floor is  1.2=1 or  -1.2=-2)
						//  this_pixel = this_pixel * (1-shift)  + left_pixel * shift    тут   0<=shift<1    или =(1-s)T+sL  или  T + s(L-T)
						// формула в _aAlphaMap   = BG+a(FG-BG)  т.е.  = amap[sbyte, BG, FG]  т.е. та же формула.
						// формула для диагонального смещения пикселя: (T+sx(L-T))  +  sy(  (U+sx(D-U)) - (T+sx(L-T))  ), где T - this_pixel, U - up_pixel, L - left_pixel, D - diagonal_pixel (Left-Up), sx - смещение по х, sy - смещение по y
					}
				}
				else // невозможный вариант, но ))
				{
					bElse = true;
				}
			}
			else // нужны поля-> реализовано только для горизонтального движения-> для диагональных - см-> подсказки здесь и выше->
			{
				//#region даём поля + неточное попадание пикселей в рамках полей
				// ex cLLI init
				if (nBGIndxPixelStartRed < nBGCropStartRed)
					nFGLineBeginningRed = nFGCropStartRed;
				else
					nFGLineBeginningRed = ((int)((float)nFGCropStartRed / cLayerInfo->nWidth_4)) * cLayerInfo->nWidth_4;
				nBgFgLinesDelta = nBGIndxLine - cLayerInfo->nCropTopLineInBG;
				// ----------
				nRowBeginingIndx = nFGLineBeginningRed;
				if (0 == ((nBgFgLinesDelta + cLayerInfo->nOffsetTop) & 1))   // -----в dvPal это та по чётности строка, которая первой должна показывааться! Т->е-> половина движения
				{
					nFGIndxRed = nFGIndxRed + cLayerInfo->nHalfDeltaPxX_4; // для влево nHalfDeltaPxX_4  <0  для вправо >0  -> Для диагональных движений надо еще и DeltaY*With прибавлять->->->
					nLeftPxIndx = nFGIndxRed - 4;
					nRowEndingIndx = nRowBeginingIndx + cLayerInfo->nWidth_4 - 4;
					if (nFGIndxRed < nRowBeginingIndx || nLeftPxIndx > nRowEndingIndx || (nFGIndxRed > nRowEndingIndx && 0 == cLayerInfo->nHalfPathShiftPositionByteX))
					{
						nFGColorRed = 0;
						nFGColorGreen = 0;
						nFGColorBlue = 0;
						nFGColorAlpha = 0;
					}
					else if (0 != cLayerInfo->nHalfPathShiftPositionByteX) //этот вариант не может быть вычислен в _aAlphaMap
					{
						// для движения влево  cLayerInfo->nHalfPathShiftPositionByte s>=0       // левый пиксель ещё в нашей строке и наш пиксель тоже
						if (nLeftPxIndx >= nRowBeginingIndx && nFGIndxRed <= nRowEndingIndx)      //  a+s(a-b)  === amap[sbyte, a, b] , если -1<s<0, 0<sbyte<255 !!
						{
							if (pLayers[nLayerIndx][nFGIndxRed + 3] == 0) // сложение с пустым пикселем несет опасность появления (проверено) чёрного цвета из него.
							{
								nFGColorRed = pLayers[nLayerIndx][nLeftPxIndx];
								nFGColorGreen = pLayers[nLayerIndx][nLeftPxIndx + 1];
								nFGColorBlue = pLayers[nLayerIndx][nLeftPxIndx + 2];
							}
							else if (pLayers[nLayerIndx][nLeftPxIndx + 3] == 0)
							{
								nFGColorRed = pLayers[nLayerIndx][nFGIndxRed];
								nFGColorGreen = pLayers[nLayerIndx][nFGIndxRed + 1];
								nFGColorBlue = pLayers[nLayerIndx][nFGIndxRed + 2];
							}
							else
							{
								nFGColorRed = pAlphaMap[pAlphaMap_info3d[cLayerInfo->nHalfPathShiftPositionByteX - 1] + pAlphaMap_info2d[pLayers[nLayerIndx][nFGIndxRed]] + pLayers[nLayerIndx][nLeftPxIndx]];
								nFGColorGreen = pAlphaMap[pAlphaMap_info3d[cLayerInfo->nHalfPathShiftPositionByteX - 1] + pAlphaMap_info2d[pLayers[nLayerIndx][nFGIndxRed + 1]] + pLayers[nLayerIndx][nLeftPxIndx + 1]];
								nFGColorBlue = pAlphaMap[pAlphaMap_info3d[cLayerInfo->nHalfPathShiftPositionByteX - 1] + pAlphaMap_info2d[pLayers[nLayerIndx][nFGIndxRed + 2]] + pLayers[nLayerIndx][nLeftPxIndx + 2]];
							}
							nFGColorAlpha = pAlphaMap[pAlphaMap_info3d[cLayerInfo->nHalfPathShiftPositionByteX - 1] + pAlphaMap_info2d[pLayers[nLayerIndx][nFGIndxRed + 3]] + pLayers[nLayerIndx][nLeftPxIndx + 3]];
						}
						else if (nLeftPxIndx < nRowBeginingIndx) // только мы в строке
						{
							nFGColorRed = pLayers[nLayerIndx][nFGIndxRed];
							nFGColorGreen = pLayers[nLayerIndx][nFGIndxRed + 1];
							nFGColorBlue = pLayers[nLayerIndx][nFGIndxRed + 2];
							nFGColorAlpha = pAlphaMap[pAlphaMap_info3d[cLayerInfo->nHalfPathShiftPositionByteX - 1] + pAlphaMap_info2d[pLayers[nLayerIndx][nFGIndxRed + 3]]];
						}
						else if (nFGIndxRed > nRowEndingIndx)  // только левый в строке
						{
							nFGColorRed = pLayers[nLayerIndx][nLeftPxIndx];
							nFGColorGreen = pLayers[nLayerIndx][nLeftPxIndx + 1];
							nFGColorBlue = pLayers[nLayerIndx][nLeftPxIndx + 2];
							nFGColorAlpha = pAlphaMap[pAlphaMap_info3d[cLayerInfo->nHalfPathShiftPositionByteX - 1] + pLayers[nLayerIndx][nLeftPxIndx + 3]];
						}
						else // невозможный вариант, но ))
						{
							bElse = true;
						}
					}
					else
					{
						nFGColorRed = pLayers[nLayerIndx][nFGIndxRed];
						nFGColorGreen = pLayers[nLayerIndx][nFGIndxRed + 1];
						nFGColorBlue = pLayers[nLayerIndx][nFGIndxRed + 2];
						nFGColorAlpha = pLayers[nLayerIndx][nFGIndxRed + 3];
					}
				}
				else if (bMainCondition)    // -----в dvPal это та по чётности строка, которая второй должна показывааться! Т.е. целое движение
				{
					if (0 != cLayerInfo->nShiftPositionByteX)
					{
						nLeftPxIndx = nFGIndxRed - 4;
						if (nLeftPxIndx >= nRowBeginingIndx) // левый пиксель ещё в нашей строке
						{
							if (pLayers[nLayerIndx][nFGIndxRed + 3] == 0) // сложение с пустым пикселем несет опасность появления (проверено) чёрного цвета из него.
							{
								nFGColorRed = pLayers[nLayerIndx][nLeftPxIndx];
								nFGColorGreen = pLayers[nLayerIndx][nLeftPxIndx + 1];
								nFGColorBlue = pLayers[nLayerIndx][nLeftPxIndx + 2];
							}
							else if (pLayers[nLayerIndx][nLeftPxIndx + 3] == 0)
							{
								nFGColorRed = pLayers[nLayerIndx][nFGIndxRed];
								nFGColorGreen = pLayers[nLayerIndx][nFGIndxRed + 1];
								nFGColorBlue = pLayers[nLayerIndx][nFGIndxRed + 2];
							}
							else
							{
								nFGColorRed = pAlphaMap[pAlphaMap_info3d[cLayerInfo->nShiftPositionByteX - 1] + pAlphaMap_info2d[pLayers[nLayerIndx][nFGIndxRed]] + pLayers[nLayerIndx][nLeftPxIndx]];
								nFGColorGreen = pAlphaMap[pAlphaMap_info3d[cLayerInfo->nShiftPositionByteX - 1] + pAlphaMap_info2d[pLayers[nLayerIndx][nFGIndxRed + 1]] + pLayers[nLayerIndx][nLeftPxIndx + 1]];
								nFGColorBlue = pAlphaMap[pAlphaMap_info3d[cLayerInfo->nShiftPositionByteX - 1] + pAlphaMap_info2d[pLayers[nLayerIndx][nFGIndxRed + 2]] + pLayers[nLayerIndx][nLeftPxIndx + 2]];
							}
							nFGColorAlpha = pAlphaMap[pAlphaMap_info3d[cLayerInfo->nShiftPositionByteX - 1] + pAlphaMap_info2d[pLayers[nLayerIndx][nFGIndxRed + 3]] + pLayers[nLayerIndx][nLeftPxIndx + 3]];
						}
						else // если наш пиксель первый в строке - он просто "ослабнет"
						{
							nFGColorRed = pLayers[nLayerIndx][nFGIndxRed];
							nFGColorGreen = pLayers[nLayerIndx][nFGIndxRed + 1];
							nFGColorBlue = pLayers[nLayerIndx][nFGIndxRed + 2];
							nFGColorAlpha = pAlphaMap[pAlphaMap_info3d[cLayerInfo->nShiftPositionByteX - 1] + pAlphaMap_info2d[pLayers[nLayerIndx][nFGIndxRed + 3]]];
						}
					}
					else
					{
						bElse = true;
						// else - присвоение в элзе было уже выше этого всего блока - просто обычный пиксель без шифта
					}
				}
				else
				{
					nFGColorRed = 0;
					nFGColorGreen = 0;
					nFGColorBlue = 0;
					nFGColorAlpha = 0;
				}
				//#endregion
			}
			//#endregion

			if (bElse)
			{
				if (bMainCondition)
				{
					nFGColorRed = pLayers[nLayerIndx][nFGIndxRed];
					nFGColorGreen = pLayers[nLayerIndx][nFGIndxRed + 1];
					nFGColorBlue = pLayers[nLayerIndx][nFGIndxRed + 2];
					nFGColorAlpha = pLayers[nLayerIndx][nFGIndxRed + 3];
				}
				else
				{
					nFGColorRed = 0;
					nFGColorGreen = 0;
					nFGColorBlue = 0;
					nFGColorAlpha = 0;
				}
			}

			nPixelAlpha = cLayerInfo->nAlphaConstant;

			if (255 == nPixelAlpha)
				nPixelAlpha = nFGColorAlpha;
			else if (0 == nFGColorAlpha)
				nPixelAlpha = 0;
			else if (0 < nPixelAlpha && 255 > nFGColorAlpha) // объединение альфы слоя с константной альфой !!!!
				nPixelAlpha = pAlphaMap2[pAlphaMap2_info2d[nFGColorAlpha - 1] + nPixelAlpha - 1];  //    (byte)(nFGColorAlpha * nPixelAlpha / 255.0 + 0.5);

			if (0 < nPixelAlpha && 0 < nExtMaskAlpha)
				nPixelAlpha = pAlphaMap3[pAlphaMap3_info2d[nPixelAlpha - 1] + nExtMaskAlpha - 1];   //(byte)(nPixelAlpha * (1 - nExtMaskAlpha / 255f) + 0.5);   [1--255;1--254(-1)]

			if (0 < nPixelAlpha)
			{
				nBGIndxAlpha = nBGIndxRed + 3;
				if (255 == nPixelAlpha || 0 == pLayers[0][nBGIndxAlpha])
				{
					pLayers[0][nBGIndxRed] = nFGColorRed;
					pLayers[0][nBGIndxRed + 1] = nFGColorGreen;
					pLayers[0][nBGIndxRed + 2] = nFGColorBlue;
				}
				else
				{                           //индекс меньше, т.к. 0-е значение альфы мы не считаем и все индексы сдвинулись...
					nPixelAlphaIndx = nPixelAlpha - 1;
					if (!bBGChanged)
					{
						pLayers[0][nBGIndxRed] = pAlphaMap[pAlphaMap_info3d[nPixelAlphaIndx] + nFGColorRed];
						pLayers[0][nBGIndxRed + 1] = pAlphaMap[pAlphaMap_info3d[nPixelAlphaIndx] + nFGColorGreen];
						pLayers[0][nBGIndxRed + 2] = pAlphaMap[pAlphaMap_info3d[nPixelAlphaIndx] + nFGColorBlue];
					}
					else
					{
						nBGIndxGreen = nBGIndxRed + 1;    //НА САМОМ ДЕЛЕ - это  BGRA , а не RGBA ))
						nBGIndxBlue = nBGIndxRed + 2;
						//pLayers[0][nBGIndxRed] = 255 < (nResult = (nPixelAlpha * (nFGColorRed - pLayers[0][nBGIndxRed])) / 255.0 + pLayers[0][nBGIndxRed] + 0.5) ? 255: (byte)nResult;   при большой нагрузке побеждают массивы - 13сек чистая математика, 12сек только альфа на массивах, 11сек всё на массивах. при маленькой - разница почти не видна.
						pLayers[0][nBGIndxRed] = pAlphaMap[pAlphaMap_info3d[nPixelAlphaIndx] + pAlphaMap_info2d[pLayers[0][nBGIndxRed]] + nFGColorRed];
						pLayers[0][nBGIndxGreen] = pAlphaMap[pAlphaMap_info3d[nPixelAlphaIndx] + pAlphaMap_info2d[pLayers[0][nBGIndxGreen]] + nFGColorGreen];
						pLayers[0][nBGIndxBlue] = pAlphaMap[pAlphaMap_info3d[nPixelAlphaIndx] + pAlphaMap_info2d[pLayers[0][nBGIndxBlue]] + nFGColorBlue];
					}
				}
				// цвета надо тушить, как выше сделано, если это всё на жестком заднике мы делаем (0,0,0,255) и альфу не трогать, а если это в воздухе всё (0,0,0,0) , то, как ниже, просто альфу надо положить!
				if (!bBGAChanged || pLayers[0][nBGIndxAlpha] < nPixelAlpha)   // очередная попытка примирить альфу с действительностью ))
				{
					pLayers[0][nBGIndxAlpha] = nPixelAlpha;
					bBGAChanged = true;
				}
				bBGChanged = true;
			}
		}
		nExtMaskAlpha = 0;   // полностью не маскируем слой
	}
	if (!bBGChanged)
	{
		pLayers[0][nBGIndxRed] = 0;
		pLayers[0][nBGIndxRed + 1] = 0;
		pLayers[0][nBGIndxRed + 2] = 0;
	}
	if (!bBGAChanged)
		pLayers[0][nBGIndxRed + 3] = 0;

}
#endif // _INC_ARRAY_H_
